//
// Created by LIUJUN on 19/1/2025.
//
#include "CANDY/GANNSIndex/GANNSIndex.h"
//#include <hip/hip_runtime.h>
namespace CANDY{

GANNSIndex::GANNSIndex() {
  graph_type = "nsw";
}

bool GANNSIndex::setConfig(INTELLI::ConfigMapPtr cfg) {
  AbstractIndex::setConfig(cfg);
  if(this->faissMetric==faiss::METRIC_INNER_PRODUCT) {
    USE_IP_DIST_=1;
    USE_L2_DIST_=0;
    USE_COS_DIST_=0;
    INTELLI_INFO("switch into inner product");
  } else if(this->faissMetric==faiss::METRIC_L2) {
      USE_IP_DIST_=0;
      USE_L2_DIST_=1;
      USE_COS_DIST_=0;
    INTELLI_INFO("switch back to L2");
  } else if(this->faissMetric==faiss::METRIC_INNER_PRODUCT) {
      USE_IP_DIST_=0;
      USE_L2_DIST_=0;
      USE_COS_DIST_=1;
    INTELLI_INFO("switch into cosin distance");
  }

  vecDim= cfg->tryI64("vecDim", 768, true);
  GANNS_DIM = vecDim;
  return true;
}
bool GANNSIndex::loadInitialTensorWithIds(std::vector<faiss::idx_t> ids, torch::Tensor &t) {
  points = new gData(t);
  graph = new NavigableSmallWorldGraphWithFixedDegree(points);
  graph->USE_L2_DIST_= USE_L2_DIST_;
  graph->USE_IP_DIST_ = USE_IP_DIST_;
  graph->USE_COS_DIST_ = USE_COS_DIST_;
  graph->Establishment(ids,64, 10);
  Ids.insert(Ids.end(), ids.begin(), ids.end());
  return true ;
}
bool GANNSIndex::insertTensorWithIds(std::vector<faiss::idx_t> ids, torch::Tensor &t) {
  points->appendPoints(t);
  graph->reset();
  Ids.insert(Ids.end(),ids.begin(),ids.end());
  graph->Establishment(Ids,64,16);
  return true;
}


std::vector<faiss::idx_t> GANNSIndex::searchIndex(torch::Tensor q, int64_t k)
{
  int *results = nullptr ;
  auto query_points = new gData(q);
  int num_of_topk = k;
  int num_of_candidates = 64;
  std::vector<faiss::idx_t> ru (q.size(0)*k);
  graph->SearchTopKonDevice(query_points->GetFirstPositionofPoint(0), num_of_topk, results, query_points->GetNumPoints(), num_of_candidates);
  for(int i = 0 ; i < query_points->GetNumPoints(); i++) {

    for(int j = 0 ; j < num_of_topk; j++) {
      int idx = results[i * num_of_topk + j];
      ru[i*k+j] = idx;

    }
  }
  return ru ;
}

}