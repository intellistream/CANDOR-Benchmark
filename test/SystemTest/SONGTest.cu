/*
* Copyright (C) 2024 by the INTELLI team
 * Created on: 25-2-12 下午7:26
 * Description: ${DESCRIPTION}
 */
#define CATCH_CONFIG_MAIN
#include "catch.hpp"
#include <CANDY/SONG/SONG.hpp>
#include <torch/torch.h>
#include <CANDY.h>

using namespace std;
using namespace INTELLI;
using namespace torch;
using namespace CANDY;

TEST_CASE("SONG: Insert and Search", "[SONG]") {
    CANDY::SONG index;
    INTELLI::ConfigMapPtr config = newConfigMap();
    config->edit("vecDim", (int64_t)4);
    config->edit("vecVolume", (int64_t)100);
    config->edit("metricType", "L2");
    REQUIRE(index.setConfig(config));

    // 生成 5 个随机 4 维张量
    torch::Tensor data = torch::rand({50, 4});
    REQUIRE(index.insertTensor(data));

    // 检查索引大小是否正确
    REQUIRE(index.size() == 50);

    // 进行搜索
    torch::Tensor query = torch::rand({1, 4});
    auto results = index.searchTensor(query, 10);

    REQUIRE(results.size() == 1);
    REQUIRE(results[0].size(0) == 10);

    // 输出搜索结果
    std::cout << "Search Results: " << results[0] << std::endl;
}
