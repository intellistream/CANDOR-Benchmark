#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024 by the INTELLI team
 * Created on: 25-2-19 下午3:32
 * Description: ${DESCRIPTION}
 */
#include <faiss/invlists/BlockInvertedListsXHS.h>
#include <faiss/impl/CodePacker.h>
#include <faiss/impl/FaissAssert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <faiss/impl/io.h>
#include <faiss/impl/io_macros.h>

namespace faiss {


BlockInvertedListsXHS::BlockInvertedListsXHS(
        size_t nlist,
        size_t vec_per_block,
        size_t block_size)
        : InvertedLists(nlist, InvertedLists::INVALID_CODE_SIZE),
          n_per_block(vec_per_block),
          block_size(block_size),
          rearrange_threshold(10000) { // 设置碎片整理的阈值
    memory_pool.resize(1UL << 30); // 预分配1GB GPU内存
    cur_pool_ptr.store(0);
    heads.resize(nlist, nullptr);
    tails.resize(nlist, nullptr);
}

BlockInvertedListsXHS::BlockInvertedListsXHS(
        size_t nlist,
        const CodePacker* packer)
        : InvertedLists(nlist, InvertedLists::INVALID_CODE_SIZE),
          n_per_block(packer->nvec),
          block_size(packer->block_size),
          packer(packer),
          rearrange_threshold(10000) { // 设置碎片整理的阈值
    memory_pool.resize(1UL << 30); // 预分配1GB GPU内存
    cur_pool_ptr.store(0);
    heads.resize(nlist, nullptr);
    tails.resize(nlist, nullptr);
}
// 分配新的 GPU 内存块
MemoryBlock* BlockInvertedListsXHS::allocate_block() {
    size_t block_mem = sizeof(MemoryBlock) + n_per_block * (sizeof(idx_t) + code_size);
    block_mem = (block_mem + 63) & ~63;
    size_t ptr = cur_pool_ptr.fetch_add(block_mem);
    FAISS_ASSERT(ptr + block_mem <= memory_pool.size());

    auto* block = reinterpret_cast<MemoryBlock*>(memory_pool.data() + ptr);
    block->prev = nullptr;
    block->next = nullptr;
    block->capacity = n_per_block;
    block->size = 0;
    block->is_merged = false;
    block->ids = reinterpret_cast<idx_t*>(block + 1);
    block->codes = reinterpret_cast<uint8_t*>(block->ids + n_per_block);
    return block;
}

// 插入向量数据
void BlockInvertedListsXHS::insert_entries(
        size_t list_no,
        size_t n_entry,
        const idx_t* ids_in,
        const uint8_t* codes) {
    FAISS_ASSERT(list_no < nlist);

    // 获取当前尾部块
    MemoryBlock* current = tails[list_no];

    // 计算单个向量的编码数据大小
    size_t single_code_size = packer ? packer->code_size : code_size;

    size_t remaining = n_entry; // 还需要插入的向量数量
    const idx_t* cur_ids = ids_in;
    const uint8_t* cur_codes = codes;

    while (remaining > 0) {
        if (!current || current->size >= current->capacity) {
            // 需要分配新块
            MemoryBlock* new_block = allocate_block();

            // 更新链表
            if (tails[list_no]) {
                tails[list_no]->next = new_block;
                new_block->prev = tails[list_no];
            } else {
                heads[list_no] = new_block;
            }
            tails[list_no] = new_block;
            current = new_block;
        }

        // 计算可以存入当前块的向量数量
        size_t space_left = current->capacity - current->size;
        size_t to_insert = std::min(space_left, remaining);

        // 逐个复制 ID 到 GPU
        hipMemcpy(current->ids + current->size, cur_ids, to_insert * sizeof(idx_t), hipMemcpyHostToDevice);

        // 逐个复制 Codes 到 GPU
        for (size_t i = 0; i < to_insert; ++i) {
            // 每次插入一个向量，调用 pack_1 来压缩代码
            if (packer) {
                FAISS_ASSERT(packer->code_size > 0);
                FAISS_ASSERT(current->codes != nullptr);
                // printf("insert:packer\n");
                // 获取目标位置
                uint8_t* dst_codes = current->codes + (current->size + i) * single_code_size;
                packer->pack_1(cur_codes + i * single_code_size, 0, dst_codes);
            } else {
                // 如果没有packer，直接复制数据
                hipMemcpy(current->codes + (current->size + i) * single_code_size,
                           cur_codes + i * single_code_size,
                           single_code_size,
                           hipMemcpyHostToDevice);
            }
        }
        // 更新计数
        current->size += to_insert;
        remaining -= to_insert;
        cur_ids += to_insert;
        cur_codes += to_insert * single_code_size;
    }

    // 检查是否触发碎片整理
    size_t total_size = 0;
    total_size = list_size(list_no);
    if (total_size >= rearrange_threshold) {
        rearrange_if_needed(list_no);
    }
}

// 碎片整理
void BlockInvertedListsXHS::rearrange_if_needed(size_t list_no) {
    MemoryBlock* current = heads[list_no];
    // printf("rearrange_if_needed\n");
    while (current) {
        if (current->is_merged) {
            current = current->next;
            continue;
        }
        MemoryBlock* next_block = current->next;
        if (next_block && current->size + next_block->size <= n_per_block) {
            hipMemcpyAsync(current->ids + current->size, next_block->ids,
                            next_block->size * sizeof(idx_t), hipMemcpyDeviceToDevice);
            hipMemcpyAsync(current->codes + current->size * code_size, next_block->codes,
                            next_block->size * code_size, hipMemcpyDeviceToDevice);
            hipStreamSynchronize(0);
            current->size += next_block->size;
            current->next = next_block->next;
            next_block->is_merged = true;  // 确保标记更新
        }
        current = current->next;
    }
}

size_t BlockInvertedListsXHS::list_size(size_t list_no) const {
    FAISS_ASSERT(list_no < nlist);
    size_t total = 0;
    MemoryBlock* p = heads[list_no];
    while (p != nullptr) {
        total += p->size;
        p = p->next;
    }
    return total;
}

const uint8_t* BlockInvertedListsXHS::get_codes(size_t list_no) const {
    FAISS_ASSERT(list_no < nlist);
    // tips:返回首个内存块的codes地址（需外部处理链表访问）
    return heads[list_no] ? heads[list_no]->codes : nullptr;
}

const idx_t* BlockInvertedListsXHS::get_ids(size_t list_no) const {
    FAISS_ASSERT(list_no < nlist);
    // tips:返回首个内存块的ids地址（同上）
    return heads[list_no] ? heads[list_no]->ids : nullptr;
}

size_t BlockInvertedListsXHS::add_entries(
        size_t list_no,
        size_t n_entry,
        const idx_t* ids,
        const uint8_t* code) {
    FAISS_ASSERT(list_no < nlist);
    // 若当前列表为空，直接插入
    if (heads[list_no] == nullptr) {
        insert_entries(list_no, n_entry, ids, code);
        return 0;
    }
    // 否则追加到尾部（需计算偏移量）
    size_t offset = list_size(list_no);
    insert_entries(list_no, n_entry, ids, code);
    return offset;
}

void BlockInvertedListsXHS::update_entries(
        size_t list_no,
        size_t offset,
        size_t n_entry,
        const idx_t* ids,
        const uint8_t* code) {
    FAISS_THROW_MSG("BlockInvertedListsXHS::update_entries not implemented");
}

void BlockInvertedListsXHS::resize(size_t list_no, size_t new_size) {
    FAISS_ASSERT(list_no < nlist);
    size_t current_size = list_size(list_no);
    if (new_size == current_size) return;

    if (new_size < current_size) {
        // 缩减：从尾部删除多余块
        MemoryBlock* p = tails[list_no];
        while (p != nullptr && current_size > new_size) {
            size_t remove_num = std::min(p->size, current_size - new_size);
            p->size -= remove_num;
            current_size -= remove_num;
            if (p->size == 0) {
                // 删除空块
                if (p->prev) p->prev->next = nullptr;
                tails[list_no] = p->prev;
                if (p == heads[list_no]) heads[list_no] = nullptr;
                MemoryBlock* to_delete = p;
                p = p->prev;
                // 注意：内存池无需释放，由预分配管理
            } else {
                p = p->prev;
            }
        }
    } else {
        // 扩展：分配新块填充0（需按block_size对齐）
        size_t add_num = new_size - current_size;
        size_t needed_blocks = (add_num + n_per_block - 1) / n_per_block;
        std::vector<idx_t> dummy_ids(add_num, -1);
        std::vector<uint8_t> dummy_codes(add_num * code_size, 0);
        // 插入填充数据
        insert_entries(list_no, add_num, dummy_ids.data(), dummy_codes.data());
    }
}

BlockInvertedListsXHS::~BlockInvertedListsXHS() {
    for (size_t i = 0; i < nlist; i++) {
        // MemoryBlock* block = heads[i];
        // while (block) {
        //     MemoryBlock* next = block->next;
        //     delete block;
        //     block = next;
        // }
        heads[i] = nullptr;
        tails[i] = nullptr;
    }
}

BlockInvertedListsXHSIOHook::BlockInvertedListsXHSIOHook()
        : InvertedListsIOHook("ilxs", typeid(BlockInvertedListsXHS).name()) {}

void BlockInvertedListsXHSIOHook::write(const InvertedLists* ils_in, IOWriter* f) const {
    uint32_t h = fourcc("ilxs");
    WRITE1(h);
    const BlockInvertedListsXHS* il = dynamic_cast<const BlockInvertedListsXHS*>(ils_in);
    WRITE1(il->nlist);
    WRITE1(il->code_size);
    WRITE1(il->n_per_block);
    WRITE1(il->block_size);

    // 逐个倒排列表存储
    for (size_t i = 0; i < il->nlist; i++) {
        size_t list_size = il->list_size(i);
        WRITE1(list_size);

        if (list_size > 0) {
            // **拷贝 GPU 端数据到 CPU**
            std::vector<idx_t> ids_cpu(list_size);
            std::vector<uint8_t> codes_cpu(list_size * il->code_size);

            hipMemcpy(ids_cpu.data(), il->get_ids(i), list_size * sizeof(idx_t), hipMemcpyDeviceToHost);
            hipMemcpy(codes_cpu.data(), il->get_codes(i), list_size * il->code_size, hipMemcpyDeviceToHost);

            // **写入文件**
            WRITEVECTOR(ids_cpu);
            WRITEVECTOR(codes_cpu);
        }
    }
}

InvertedLists* BlockInvertedListsXHSIOHook::read(IOReader* f, int /* io_flags */) const {
    BlockInvertedListsXHS* il = new BlockInvertedListsXHS();
    READ1(il->nlist);
    READ1(il->code_size);
    READ1(il->n_per_block);
    READ1(il->block_size);

    il->heads.resize(il->nlist, nullptr);
    il->tails.resize(il->nlist, nullptr);

    for (size_t i = 0; i < il->nlist; i++) {
        size_t list_size;
        READ1(list_size);

        if (list_size > 0) {
            // **从文件读取数据**
            std::vector<idx_t> ids_cpu(list_size);
            std::vector<uint8_t> codes_cpu(list_size * il->code_size);
            READVECTOR(ids_cpu);
            READVECTOR(codes_cpu);

            // **分配 GPU 内存并拷贝数据**
            MemoryBlock* block = il->allocate_block();
            hipMemcpy(block->ids, ids_cpu.data(), list_size * sizeof(idx_t), hipMemcpyHostToDevice);
            hipMemcpy(block->codes, codes_cpu.data(), list_size * il->code_size, hipMemcpyHostToDevice);
            block->size = list_size;

            il->heads[i] = block;
            il->tails[i] = block;
        }
    }

    return il;
}
} // namespace faiss
